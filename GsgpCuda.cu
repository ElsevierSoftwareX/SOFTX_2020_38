#include "hip/hip_runtime.h"
/*<one line to give the program's name and a brief idea of what it does.>
    Copyright (C) 2020 José Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

     This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU Affero General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//! \file   GsgpCuda.cu
//! \brief  file containing the main with the geometric semantic genetic programming algorithm
//! \author Jose Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith
//! \date   created on 25/01/2020
#include "GsgpCuda.cpp"

/*!
* \fn       int main(int argc, const char **argv)
* \brief    main method that runs the GSGP algorithm and test the best model generate by GSGP-CUDA
* \param    int argc: number of parameters of the program
* \param    const char **argv: array of strings that contains the parameters of the program
* \return   int: 0 if the program ends without errors
* \date     25/01/2020
* \author   Jose Manuel Muñoz Contreras, Leonardo Trujillo, Daniel E. Hernandez, Perla Juárez Smith
* \file     GsgpCuda.cu
*/
int main(int argc, char **argv){

    printf("\n Starting GsgpCuda \n\n");
    
    srand(time(NULL)); /*!< Initialization of the seed for the generation of random numbers*/

    readConfigFile(&config); /*!< reading the parameters of the algorithm */

    hipSetDevice(0); /*!< Select a GPU device*/
    
    char trainFile[50]="";    /*!< Name of the train file*/
    char testFile[50]="";     /*!< Name of the test file*/
    char output_model[50]=""; /*!< Name of output files*/
    char pathTrace[50]="";    /*!< Name of the file trace of best model*/
    char path_test[50]="";    /*!< Name of the file with unsseen test instances*/
    char pathOutFile[50]="";  /*!< Name of the file to output values*/
    for (int i=1; i<argc-1; i++){
        if(strncmp(argv[i],"-train_file",10) == 0) {
            strcat(trainFile,argv[++i]);
        }else if (strncmp(argv[i],"-test_file",10) == 0) {
            strcat(testFile,argv[++i]);
        }else if (strncmp(argv[i],"-output_model",10)==0) {
            strcat(output_model,argv[++i]);
        }else if (strncmp(argv[i],"-model",10)==0) {
            strcat(pathTrace,argv[++i]);
        }else if (strncmp(argv[i],"-input_data",10)==0) {
            strcat(path_test,argv[++i]);
        }else if (strncmp(argv[i],"-prediction_output",10)==0) {
            strcat(pathOutFile,argv[++i]);
        }     
    }

    std::string trainFile_s(trainFile);
 
    std::string testFile_s(testFile);
 
    std::string outputNameFiles(output_model); ///*!< Name of file for save the output files*/
 
    std::string la (pathTrace);
 
    std::string logPath (config.logPath); /* Path of directory for data files and log files generated in execution */
 
    std::string namePopulation = "_initialPopulation"; /*!< Name of file for save the initial population  */

    namePopulation = outputNameFiles + namePopulation;

    std::string nameRandomTrees = "_randomTrees"; /*!< name of file for save the random trees */
    
    nameRandomTrees = outputNameFiles + nameRandomTrees; 
        
    if (!trainFile_s.empty() && testFile_s.empty()){
        
        countInputFile(trainFile, nrow, nvar); ///Counting the number of rows and variables of the train file
        
        nvar--; 

        individualLength = config.maxIndividualLength; /*!< Variable that stores maximum depth for individuals */

        sizeMemIndividuals = sizeof(float) * config.populationSize; /*!< Variable that stores size in bytes of the number of individuals in the initial population*/

        twoSizeMemPopulation = sizeof(float) * (config.populationSize*2); /*!< Variable that stores twice the size in bytes of an initial population to store random numbers*/

        sizeMemPopulation = sizeof(float) * config.populationSize * individualLength; /*!< Variable that stores size in bytes for initial population*/
        
        twoSizePopulation = (config.populationSize*2); /*!< Variable storing twice the initial population of individuals to generate random positions*/

        sizeMemSemanticTrain = sizeof(float)*(config.populationSize*nrow); /*!< Variable that stores the size in bytes of semantics for the entire population with training data*/

        sizeMemDataTrain = sizeof(float)*(nrow*nvar); /*!< Variable that stores the size in bytes the size of the training data*/

        sizeElementsSemanticTrain = (config.populationSize*nrow); /*!< Variable that stores training data elements*/

        vectorTracesMem = (sizeof(entry_)*config.numberGenerations*config.populationSize); /*!< Variable that stores the size in bytes of the structure to store the survival record*/

        std::string logPath (config.logPath); /* Path of directory for data files and log files generated in execution */

        std::string namePopulation = "_initialPopulation"; /*!< Name of file for save the initial population  */

        namePopulation = outputNameFiles + namePopulation;

        std::string nameRandomTrees = "_randomTrees"; /*!< name of file for save the random trees */
        
        nameRandomTrees = outputNameFiles + nameRandomTrees;

        /* Check if log and data diectories exists */
        checkDirectoryPath(logPath);
        
        float executionTime = 0, initialitionTimePopulation = 0, timeComputeSemantics = 0, generationTime = 0; /*!< Variables that store the time in milliseconds between the events mark1 and mark2.*/

        std::string timeExecution1 = "_processing_time"; /*!< Variable name structure responsible for indicating the run*/
        std::string timeExecution2 = ".csv"; /*!< Variable name structure responsible for indicating the file extension*/
        timeExecution1 = logPath + outputNameFiles + timeExecution1 + timeExecution2; /*!< Variable that stores file name matching*/
        std::ofstream times(timeExecution1,ios::out); /*!< pointer to the timeExecution1 file that contains the time consumed by the different algorithm modules*/
 
        hipEvent_t startRun, stopRun;  /*!< Variable used to create a start mark and a stop mark to create events*/
        hipEventCreate(&startRun);     /*!< function that initializes the start event*/
        hipEventCreate(&stopRun);      /*!< function that initializes the stop event*/

        hiprandState_t* states; /*!< CUDA's random number library uses hiprandState_t to keep track of the seed value we will store a random state for every thread*/
        hipMalloc((void**) &states, config.populationSize * sizeof(hiprandState_t)); /*!< allocate space on the GPU for the random states*/
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/

        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/
        init<<<gridSize, blockSize>>>(time(0), states); /*!< invoke the GPU to initialize all of the random states*/

        hipEventRecord(startRun);     
        std::string fitnessTrain  = "_fitnestrain"; /**/
        std::string fitnessTrain2 = ".csv"; /**/
        fitnessTrain = logPath + outputNameFiles + fitnessTrain + fitnessTrain2; /**/
        std::ofstream fitTraining(fitnessTrain,ios::out); /*!< pointer to the file fitnesstrain.csv containing the training fitness of the best individual at each generation*/

        hipblasHandle_t handle; /*!< the handle to the cuBLAS library context*/
        hipblasCreate(&handle); /*!< initialized using the function and is explicitly passed to every subsequent library function call*/
 
        hInitialPopulation = (float *)malloc(sizeMemPopulation); 
        hRandomTrees = (float *)malloc(sizeMemPopulation); 
        checkCudaErrors(hipMalloc((void **)&dRandomTrees, sizeMemPopulation)); 
        checkCudaErrors(hipMalloc((void **)&dInitialPopulation, sizeMemPopulation));
        checkCudaErrors(hipMallocManaged(&vectorTraces,vectorTracesMem));
        checkCudaErrors(hipMallocManaged(&uDataTrain, sizeMemDataTrain));     
        checkCudaErrors(hipMallocManaged(&uDataTrainTarget, sizeof(float)*nrow));            
        checkCudaErrors(hipMallocManaged(&uFit, sizeMemIndividuals));  
        checkCudaErrors(hipMallocManaged(&uSemanticTrainCases,sizeMemSemanticTrain));       
        checkCudaErrors(hipMallocManaged(&uSemanticRandomTrees,sizeMemSemanticTrain));      
        checkCudaErrors(hipMalloc((void**)&uPushGenes, sizeMemIndividuals));
        checkCudaErrors(hipMalloc((void**)&uStackInd, sizeMemPopulation));            
       
        readInpuDataTrain(trainFile, uDataTrain, uDataTrainTarget, nrow, nvar); /// load set data train **/
        
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/            
        
        hipEvent_t startInitialPop, stopInitialPop; /*!< this section declares and initializes the Variables for the events and captures the time elapsed in the initialization of the initial population in the GPU*/
        hipEventCreate(&startInitialPop);
        hipEventCreate(&stopInitialPop);
        hipEventRecord(startInitialPop);

        ///invokes the GPU to initialize the initial population
        initializePopulation<<< gridSize, blockSize >>>(dInitialPopulation, nvar, individualLength, states, config.maxRandomConstant, 4, config.functionRatio, config.variableRatio);
        cudaErrorCheck("initializePopulation");

        hipEventRecord(stopInitialPop);
        hipEventSynchronize(stopInitialPop);
        hipEventElapsedTime(&initialitionTimePopulation, startInitialPop, stopInitialPop);
        hipEventDestroy(startInitialPop);
        hipEventDestroy(stopInitialPop);    
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hInitialPopulation, dInitialPopulation, sizeMemPopulation, hipMemcpyDeviceToHost);    
        saveIndividuals(logPath,hInitialPopulation, namePopulation, individualLength,config.populationSize);  

        ///*!< invokes the GPU to initialize the random trees*/
        initializePopulation<<< gridSize, blockSize >>>(dRandomTrees, nvar, individualLength, states, config.maxRandomConstant,4,config.functionRatio, config.variableRatio);    
        cudaErrorCheck("initializePopulation");    
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hRandomTrees, dRandomTrees,sizeMemPopulation, hipMemcpyDeviceToHost);    
        saveIndividuals(logPath,hRandomTrees, nameRandomTrees,individualLength,config.populationSize); 

        hipEvent_t startComputeSemantics, stopComputeSemantics; /*!< This section declares and initializes the Variables for the events and captures the time elapsed in the interpretation of the initial population in the GPU*/
        hipEventCreate(&startComputeSemantics);
        hipEventCreate(&stopComputeSemantics);
        hipEventRecord(startComputeSemantics);    
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemantics, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/      
     
        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticTrainCases, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");            
        /*!< invokes the GPU to interpret the random trees with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticRandomTrees, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");            
        hipEventRecord(stopComputeSemantics);
        hipEventSynchronize(stopComputeSemantics);
        hipEventElapsedTime(&timeComputeSemantics, startComputeSemantics, stopComputeSemantics);
        hipEventDestroy(startComputeSemantics);
        hipEventDestroy(stopComputeSemantics);  

        /*!< memory is deallocated for training data and auxiliary vectors for the interpreter*/
        hipFree(uDataTrain);
        hipFree(uStackInd);
        hipFree(uPushGenes);  

        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
        gridSize = (config.populationSize + blockSize - 1) / blockSize;         
        
        /*!< invokes the GPU to calculate the error (RMSE) the initial population*/
        computeError<<< gridSize, blockSize >>>(uSemanticTrainCases, uDataTrainTarget, uFit, nrow);
        cudaErrorCheck("computeError");                   
        
        
        hipblasIsamin(handle, config.populationSize, uFit, incx1, &result);
        indexBestIndividual = result-1;
        
        /*!< function is necessary so that the CPU does not continue with the execution of the program and allows to capture the fitness*/
        hipDeviceSynchronize();
        
        /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
        fitTraining << 0 << "," <<uFit[indexBestIndividual]<<endl;

        checkCudaErrors(hipMallocManaged(&uSemanticTrainCasesNew,sizeMemSemanticTrain));
        checkCudaErrors(hipMallocManaged(&uFitNew, sizeMemPopulation));
        
        hipEvent_t startGsgp, stopGsgp;
        hipEventCreate(&startGsgp);
        hipEventCreate(&stopGsgp);          
        
        hiprandState_t* State;
        hipMalloc((void**) &State, (twoSizePopulation) * sizeof(hiprandState_t));
        checkCudaErrors(hipMallocManaged(&indexRandomTrees,twoSizeMemPopulation));
        checkCudaErrors(hipMallocManaged(&mutationStep,sizeMemPopulation)); 
        hiprandState_t* statesMutationStep;
        hipMalloc((void**) &statesMutationStep, (sizeMemPopulation) * sizeof(hiprandState_t));
        int index =0;   
        
        /*!< main GSGP cycle*/
        for ( int generation=1; generation<=config.numberGenerations; generation++){

            /*!< register execution time*/
            hipEventRecord(startGsgp);
            gridSize =0, blockSize=0;
            index = generation-1;
            
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;
            init<<<gridSize, blockSize>>>(time(NULL)*index, State); /*!< initializes the random number generator*/
            cudaErrorCheck("init");

            /*!< invokes the GPU to initialize the random positions of the random trees*/
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeIndexRandomTrees, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;

            initializeIndexRandomTrees<<<gridSize,blockSize >>>( config.populationSize, indexRandomTrees, State );
            cudaErrorCheck("initializeIndexRandomTrees");

            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize);
            gridSize = (config.populationSize + blockSize - 1) / blockSize;
            init<<<gridSize, blockSize>>>(time(NULL)*index, statesMutationStep); /*!< initializes the random number generator*/
            cudaErrorCheck("init");
            
            /*!< invokes the GPU to initialize the random positions of the random trees*/
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeMutationStep, 0, config.populationSize);
            gridSize = (config.populationSize + blockSize - 1) / blockSize;

            initializeMutationStep<<<gridSize,blockSize >>>(mutationStep, statesMutationStep);
            cudaErrorCheck("initializeMutationStep");

            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, geometricSemanticMutation, 0, sizeElementsSemanticTrain);
            gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
            
           if((gridSize*blockSize)>sizeElementsSemanticTrain){
               blockSize = minGridSize;
               gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
            }

            /*!< geometric semantic mutation with semantic train*/
            geometricSemanticMutation<<< gridSize, blockSize >>>(uSemanticTrainCases, uSemanticRandomTrees,uSemanticTrainCasesNew,
            config.populationSize, nrow, sizeElementsSemanticTrain, generation, indexRandomTrees, vectorTraces, index, mutationStep);
            cudaErrorCheck("geometricSemanticMutation");
            
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
            gridSize = (config.populationSize + blockSize - 1) / blockSize;
              
            /*!< invokes the GPU to calculate the error (RMSE) the new population*/
            computeError<<< gridSize,blockSize >>>(uSemanticTrainCasesNew, uDataTrainTarget, uFitNew, nrow);
            cudaErrorCheck("computeError");
         
            hipblasIsamin(handle, config.populationSize, uFitNew, incxBestOffspring, &resultBestOffspring);
            indexBestOffspring = resultBestOffspring-1;
            hipblasIsamax(handle, config.populationSize, uFitNew, incxWorst, &resultWorst);
            indexWorstOffspring = resultWorst-1;

            /*!< set byte values*/
            hipMemset(indexRandomTrees,0,twoSizeMemPopulation);
            hipMemset(mutationStep,0,sizeMemPopulation);
            hipDeviceSynchronize();
         
            /*!< this section performs survival by updating the semantic and fitness vectors respectively*/
            if(uFitNew[indexBestOffspring] > uFit[indexBestIndividual]){
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].firstParent = indexBestIndividual;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].secondParent = -1;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].number = indexBestIndividual;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].event = -1;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].newIndividual = indexBestIndividual;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].mark= 0;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].mutStep = 0;

                for (int i = 0; i < nrow; ++i){
                    uSemanticTrainCasesNew[indexWorstOffspring*nrow+i] = uSemanticTrainCases[indexBestIndividual*nrow+i];
                }

                uFitNew[indexWorstOffspring] = uFit[indexBestIndividual];
                tempFitnes = uFit;
                uFit = uFitNew;
                uFitNew = tempFitnes;
                tempSemantic = uSemanticTrainCases;
                uSemanticTrainCases = uSemanticTrainCasesNew;
                uSemanticTrainCasesNew = tempSemantic;

                
                indexBestIndividual = indexWorstOffspring;
            }else{
                tempFitnes = uFit;
                uFit = uFitNew;
                uFitNew = tempFitnes;
                tempSemantic = uSemanticTrainCases;
                uSemanticTrainCases = uSemanticTrainCasesNew;
                uSemanticTrainCasesNew = tempSemantic;
                indexBestIndividual = indexBestOffspring;
            }
            /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
            fitTraining << generation << "," <<uFit[indexBestIndividual]<<endl;
            hipEventRecord(stopGsgp);
            hipEventSynchronize(stopGsgp);
            hipEventElapsedTime(&generationTime, startGsgp, stopGsgp);    
        }
        markTracesGeneration(vectorTraces, config.populationSize, config.numberGenerations,  indexBestIndividual);
        saveTrace(outputNameFiles,logPath, vectorTraces, config.numberGenerations, config.populationSize);
            
        /*!< at the end of the execution  to deallocate memory*/
        hipFree(indexRandomTrees);
        hipFree(vectorTraces);
        hipblasDestroy(handle);
        hipFree(dInitialPopulation);
        hipFree(dRandomTrees);
        free(hInitialPopulation);
        free(hRandomTrees);
        hipFree(uDataTrainTarget);
        hipFree(uFit);
        hipFree(uFitNew);
        hipFree(uSemanticTrainCases);
        hipFree(uSemanticRandomTrees);
        hipFree(uSemanticTrainCasesNew);
        hipFree(mutationStep);
        hipEventRecord(stopRun);
        hipEventSynchronize(stopRun);
        hipEventElapsedTime(&executionTime, startRun, stopRun);

         /*!< writing the time execution for stages the algorithm*/
        times << config.populationSize
        << "," << individualLength 
        << "," << nrow 
        << "," << nvar 
        << "," << executionTime/1000
        << "," << initialitionTimePopulation/1000
        << "," << timeComputeSemantics/1000
        << "," << generationTime/1000
        <<endl;
        hipFree(State);
        hipFree(states);
        hipFree(statesMutationStep);
        /*!< all device allocations are removed*/
        hipDeviceReset();
        return 0;
    }else if (!trainFile_s.empty() && !testFile_s.empty()) {
        /*!< this section is for the case when the user wants to run the algorithm with the training and test files*/

        countInputFile(trainFile, nrow, nvar);

        countInputFile(testFile, nrowTest, nvar);

        nvar--;

        individualLength = config.maxIndividualLength; /*!< Variable that stores maximum depth for individuals */
        
        sizeMemPopulation = sizeof(float) * config.populationSize * individualLength; /*!< Variable that stores size in bytes for initial population*/
        
        twoSizeMemPopulation = sizeof(float) * (config.populationSize*2); /*!< Variable that stores twice the size in bytes of an initial population to store random numbers*/
        
        sizeMemIndividuals = sizeof(float) * config.populationSize; /*!< Variable that stores size in bytes of the number of individuals in the initial population*/
        
        twoSizePopulation = (config.populationSize*2); /*!< Variable storing twice the initial population of individuals to generate random positions*/
        
        sizeMemSemanticTrain = sizeof(float)*(config.populationSize*nrow); /*!< Variable that stores the size in bytes of semantics for the entire population with training data*/
        
        sizeMemSemanticTest = sizeof(float)*(config.populationSize*nrowTest); /*!< Variable that stores the size in bytes of semantics for the entire population with test data*/
        
        sizeMemDataTrain = sizeof(float)*(nrow*nvar); /*!< Variable that stores the size in bytes the size of the training data*/
        
        sizeMemDataTest = sizeof(float)*(nrowTest*nvar); /*!< Variable that stores the size in bytes the size of the test data*/
        
        sizeElementsSemanticTrain = (config.populationSize*nrow); /*!< Variable that stores training data elements*/
        
        sizeElementsSemanticTest = (config.populationSize*nrowTest); /*!< Variable that stores test data elements*/
        
        vectorTracesMem = (sizeof(entry_)*(config.numberGenerations*config.populationSize)); /*!< Variable that stores the size in bytes of the structure to store the survival record*/
        
        /* Check if log and data diectories exists */
        checkDirectoryPath(logPath);
        
        float executionTime = 0, initialitionTimePopulation = 0, timeComputeSemantics = 0, generationTime = 0; /*!< Variables that store the time in milliseconds between the events mark1 and mark2.*/

        std::string timeExecution1 = "_processing_time"; /*!< Variable name structure responsible for indicating the run*/
        std::string timeExecution2 = ".csv"; /*!< Variable name structure responsible for indicating the file extension*/
        timeExecution1 = logPath + outputNameFiles + timeExecution1 + timeExecution2; /*!< Variable that stores file name matching*/
        std::ofstream times(timeExecution1,ios::out); /*!< pointer to the timeExecution1 file that contains the time consumed by the different algorithm modules*/
    
        hipEvent_t startRun, stopRun; /*!< Variable used to create a start mark and a stop mark to create events*/
        hipEventCreate(&startRun); /*!< function that initializes the start event*/
        hipEventCreate(&stopRun); /*!< function that initializes the stop event*/

        hiprandState_t* states; /*!< CUDA's random number library uses hiprandState_t to keep track of the seed value we will store a random state for every thread*/
        hipMalloc((void**) &states, config.populationSize * sizeof(hiprandState_t)); /*!< allocate space on the GPU for the random states*/
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/

        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/
        init<<<gridSize, blockSize>>>(time(0), states); /*!< invoke the GPU to initialize all of the random states*/

        hipEventRecord(startRun);     
        std::string fitnessTrain  = "_fitnestrain"; /**/
        std::string fitnessTrain2 = ".csv"; /**/
        fitnessTrain = logPath + outputNameFiles + fitnessTrain + fitnessTrain2; /**/
        std::ofstream fitTraining(fitnessTrain,ios::out); /*!< pointer to the file fitnesstrain.csv containing the training fitness of the best individual at each generation*/

        std::string fitnessTest  = "_fitnestest"; /**/
        std::string fitnessTest2 = ".csv"; /**/
        fitnessTest = logPath + outputNameFiles + fitnessTest + fitnessTest2; /**/
        std::ofstream fitTesting(fitnessTest,ios::out); /*!< pointer to the file fitnesstest.csv containing the test fitness of the best individual at each generation*/

        hipblasHandle_t handle; /*!< the handle to the cuBLAS library context*/
        hipblasCreate(&handle); /*!< initialized using the function and is explicitly passed to every subsequent library function call*/
    
        hInitialPopulation = (float *)malloc(sizeMemPopulation); 
        hRandomTrees = (float *)malloc(sizeMemPopulation); 
        checkCudaErrors(hipMalloc((void **)&dRandomTrees, sizeMemPopulation)); 
        checkCudaErrors(hipMalloc((void **)&dInitialPopulation, sizeMemPopulation));
        checkCudaErrors(hipMallocManaged(&vectorTraces,vectorTracesMem));
        checkCudaErrors(hipMallocManaged(&uDataTrain, sizeMemDataTrain));
        checkCudaErrors(hipMallocManaged(&uDataTest, sizeMemDataTest));      
        checkCudaErrors(hipMallocManaged(&uDataTrainTarget, sizeof(float)*nrow));
        checkCudaErrors(hipMallocManaged(&uDataTestTarget, sizeof(float)*nrowTest));            
        checkCudaErrors(hipMallocManaged(&uFit, sizeMemIndividuals));
        checkCudaErrors(hipMallocManaged(&uFitTest, sizeMemIndividuals));    
        checkCudaErrors(hipMallocManaged(&uSemanticTrainCases,sizeMemSemanticTrain));       
        checkCudaErrors(hipMallocManaged(&uSemanticTestCases,sizeMemSemanticTest));       
        checkCudaErrors(hipMallocManaged(&uSemanticRandomTrees,sizeMemSemanticTrain));      
        checkCudaErrors(hipMallocManaged(&uSemanticTestRandomTrees,sizeMemSemanticTest));
        checkCudaErrors(hipMalloc((void**)&uPushGenes, sizeMemIndividuals));
        checkCudaErrors(hipMalloc((void**)&uStackInd, sizeMemPopulation));            
        
        readInpuData(trainFile, testFile, uDataTrain, uDataTest, uDataTrainTarget, uDataTestTarget, nrow, nvar, nrowTest, nvar); /*!< load set data train and test*/            
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/            
        
        hipEvent_t startInitialPop, stopInitialPop; /*!< this section declares and initializes the Variables for the events and captures the time elapsed in the initialization of the initial population in the GPU*/
        hipEventCreate(&startInitialPop);
        hipEventCreate(&stopInitialPop);
        hipEventRecord(startInitialPop);

        ///invokes the GPU to initialize the initial population
        initializePopulation<<< gridSize, blockSize >>>(dInitialPopulation, nvar, individualLength, states, config.maxRandomConstant,4, config.functionRatio, config.variableRatio);
        cudaErrorCheck("initializePopulation");

        hipEventRecord(stopInitialPop);
        hipEventSynchronize(stopInitialPop);
        hipEventElapsedTime(&initialitionTimePopulation, startInitialPop, stopInitialPop);
        hipEventDestroy(startInitialPop);
        hipEventDestroy(stopInitialPop);    
        
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hInitialPopulation, dInitialPopulation, sizeMemPopulation, hipMemcpyDeviceToHost);    
        saveIndividuals(logPath,hInitialPopulation, namePopulation, individualLength,config.populationSize);  
        
        ///*!< invokes the GPU to initialize the random trees*/
        initializePopulation<<< gridSize, blockSize >>>(dRandomTrees, nvar, individualLength, states, config.maxRandomConstant,4,config.functionRatio, config.variableRatio);    
        cudaErrorCheck("initializePopulation");    
        
        ///*!<return the initial population of the device to the host*/
        hipMemcpy(hRandomTrees, dRandomTrees,sizeMemPopulation, hipMemcpyDeviceToHost);    
        saveIndividuals(logPath,hRandomTrees, nameRandomTrees,individualLength,config.populationSize);  
        
        hipEvent_t startComputeSemantics, stopComputeSemantics; /*!< This section declares and initializes the Variables for the events and captures the time elapsed in the interpretation of the initial population in the GPU*/
        hipEventCreate(&startComputeSemantics);
        hipEventCreate(&stopComputeSemantics);
        hipEventRecord(startComputeSemantics);    
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemantics, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/    
         
        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticTrainCases, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");            
        /*!< invokes the GPU to interpret the random trees with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticRandomTrees, individualLength, uDataTrain, nrow, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");            
        hipEventRecord(stopComputeSemantics);
        hipEventSynchronize(stopComputeSemantics);
        hipEventElapsedTime(&timeComputeSemantics, startComputeSemantics, stopComputeSemantics);
        hipEventDestroy(startComputeSemantics);
        hipEventDestroy(stopComputeSemantics);            
        
        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticTestCases, individualLength, uDataTest, nrowTest, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");
        
        /*!< invokes the GPU to interpret the random trees with data test*/
        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticTestRandomTrees, individualLength, uDataTest, nrowTest, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");
        hipDeviceSynchronize();
       
        /*!< memory is deallocated for training data and auxiliary vectors for the interpreter*/
        hipFree(uDataTrain);
        hipFree(uDataTest);
        hipFree(uStackInd);
        hipFree(uPushGenes);            
        
        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 
        gridSize = (config.populationSize + blockSize - 1) / blockSize;         
        
        /*!< invokes the GPU to calculate the error (RMSE) the initial population*/
        computeError<<< gridSize, blockSize >>>(uSemanticTrainCases, uDataTrainTarget, uFit, nrow);
        cudaErrorCheck("computeError");                   
        
        hipblasIsamin(handle, config.populationSize, uFit, incx1, &result);
        indexBestIndividual = result-1;

        /*!< invokes the GPU to calculate the error (RMSE) the initial population*/
        computeError<<< gridSize, blockSize >>>(uSemanticTestCases, uDataTestTarget, uFitTest, nrowTest);    
        cudaErrorCheck("computeError");         
        /*!< function is necessary so that the CPU does not continue with the execution of the program and allows to capture the fitness*/
        hipDeviceSynchronize();
        
        /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
        fitTraining << 0 << "," <<uFit[indexBestIndividual]<<endl;
        /*!< writing the  test fitness of the best individual on the file fitnesstest.csv*/
        fitTesting << 0 << "," <<uFitTest[indexBestIndividual]<<endl;              

        checkCudaErrors(hipMallocManaged(&uSemanticTrainCasesNew,sizeMemSemanticTrain));
        checkCudaErrors(hipMallocManaged(&uFitNew, sizeMemPopulation));
        checkCudaErrors(hipMallocManaged(&uSemanticTestCasesNew,sizeMemSemanticTest));
        checkCudaErrors(hipMallocManaged(&uFitTestNew, sizeMemPopulation));

        hipEvent_t startGsgp, stopGsgp;
        hipEventCreate(&startGsgp);
        hipEventCreate(&stopGsgp);          
        hiprandState_t* State;
        hipMalloc((void**) &State, (twoSizePopulation) * sizeof(hiprandState_t));
        checkCudaErrors(hipMallocManaged(&indexRandomTrees,twoSizeMemPopulation));
        checkCudaErrors(hipMallocManaged(&mutationStep,sizeMemPopulation)); 
        hiprandState_t* statesMutationStep;
        hipMalloc((void**) &statesMutationStep, (sizeMemPopulation) * sizeof(hiprandState_t));
        int index =0;       
        /*!< main GSGP cycle*/
        for ( int generation=1; generation<=config.numberGenerations; generation++){
            /*!< register execution time*/
            hipEventRecord(startGsgp);
            gridSize =0, blockSize=0;
            index = generation-1;

            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;
            init<<<gridSize, blockSize>>>(time(NULL)*index, State); /*!< initializes the random number generator*/
            cudaErrorCheck("init");

            /*!< invokes the GPU to initialize the random positions of the random trees*/
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeIndexRandomTrees, 0, twoSizePopulation);
            gridSize = (twoSizePopulation + blockSize - 1) / blockSize;
            initializeIndexRandomTrees<<<gridSize,blockSize >>>( config.populationSize, indexRandomTrees, State );
            cudaErrorCheck("initializeIndexRandomTrees");

            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, init, 0, config.populationSize);
            gridSize = (config.populationSize + blockSize - 1) / blockSize;
            init<<<gridSize, blockSize>>>(time(NULL)*index, statesMutationStep); /*!< initializes the random number generator*/
            cudaErrorCheck("init");
            
            /*!< invokes the GPU to initialize the random positions of the random trees*/
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, initializeMutationStep, 0, config.populationSize);
            gridSize = (config.populationSize + blockSize - 1) / blockSize;
            //printf("grid %i blocksize %i \n", gridSize, blockSize);
            initializeMutationStep<<<gridSize,blockSize >>>(mutationStep, statesMutationStep);
            cudaErrorCheck("initializeMutationStep");
            
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, geometricSemanticMutation, 0, sizeElementsSemanticTrain); 
            gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
            if((gridSize*blockSize)>sizeElementsSemanticTrain){
                blockSize = minGridSize;
                gridSize = (sizeElementsSemanticTrain + blockSize - 1) / blockSize;
            }
 
            /*!< geometric semantic mutation with semantic train*/
            geometricSemanticMutation<<< gridSize, blockSize >>>(uSemanticTrainCases, uSemanticRandomTrees,uSemanticTrainCasesNew,
            config.populationSize, nrow, sizeElementsSemanticTrain, generation, indexRandomTrees, vectorTraces, index, mutationStep);
            cudaErrorCheck("geometricSemanticMutation");
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeError, 0, config.populationSize); 

            gridSize = (config.populationSize + blockSize - 1) / blockSize;
            /*!< invokes the GPU to calculate the error (RMSE) the new population*/
            computeError<<< gridSize,blockSize >>>(uSemanticTrainCasesNew, uDataTrainTarget, uFitNew, nrow);
            cudaErrorCheck("computeError");
                        /*!< this section makes use of the isamin de cublas function to determine the position of the best individual of the new population*/
            
            hipblasIsamin(handle, config.populationSize, uFitNew, incxBestOffspring, &resultBestOffspring);
            indexBestOffspring = resultBestOffspring-1;
                        /*!< this section makes use of the isamin de cublas function to determine the position of the worst individual of the new population*/
            
            hipblasIsamax(handle, config.populationSize, uFitNew, incxWorst, &resultWorst);
            indexWorstOffspring = resultWorst-1;

            /*!< geometric semantic mutation with semantic test*/
            hipOccupancyMaxPotentialBlockSize(&minGridSizeTest, &blockSizeTest, geometricSemanticMutation, 0, sizeElementsSemanticTest); 
            gridSizeTest = (sizeElementsSemanticTest + blockSizeTest - 1) / blockSizeTest;
            if((gridSizeTest*blockSizeTest)>sizeElementsSemanticTest){
                blockSize = minGridSizeTest;
                gridSize = (sizeElementsSemanticTest + blockSize - 1) / blockSize;
            }
 
            geometricSemanticMutation<<< gridSizeTest, blockSizeTest >>>(uSemanticTestCases, uSemanticTestRandomTrees,uSemanticTestCasesNew,
            config.populationSize, nrowTest, sizeElementsSemanticTest, generation, indexRandomTrees, vectorTraces,index, mutationStep);
            cudaErrorCheck("geometricSemanticMutation");
            hipDeviceSynchronize();
           
            hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizeTest, computeError, 0, config.populationSize); 
            gridSizeTest = (config.populationSize + blockSizeTest - 1) / blockSizeTest;
            /*!< invokes the GPU to calculate the error (RMSE) the new population*/
            computeError<<< gridSizeTest,blockSizeTest >>>(uSemanticTestCasesNew, uDataTestTarget, uFitTestNew, nrowTest);
            cudaErrorCheck("computeError");
                        /*!< set byte values*/
            hipMemset(indexRandomTrees,0,twoSizeMemPopulation);
            hipMemset(mutationStep,0,sizeMemPopulation);
            hipDeviceSynchronize();

            /*!< this section performs survival by updating the semantic and fitness vectors respectively*/
            if(uFitNew[indexBestOffspring] > uFit[indexBestIndividual]){
                hipDeviceSynchronize();
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].firstParent = indexWorstOffspring;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].secondParent = -1;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].number = indexBestIndividual;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].event = -1;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].newIndividual = -1;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].mark = 0;
                vectorTraces[(index*config.populationSize)+indexWorstOffspring].mutStep = 0;

                for (int i = 0; i < nrow; ++i){
                    uSemanticTrainCasesNew[indexWorstOffspring*nrow+i] = uSemanticTrainCases[indexBestIndividual*nrow+i];
                }

                uFitNew[indexWorstOffspring] = uFit[indexBestIndividual];
                tempFitnes = uFit;
                uFit = uFitNew;
                uFitNew = tempFitnes;
                tempSemantic = uSemanticTrainCases;
                uSemanticTrainCases = uSemanticTrainCasesNew;
                uSemanticTrainCasesNew = tempSemantic;
                
                for (int j = 0; j < nrowTest; ++j){
                    uSemanticTestCasesNew[indexWorstOffspring*nrowTest+j] = uSemanticTestCases[indexBestIndividual*nrowTest+j];
                }
                uFitTestNew[indexWorstOffspring] = uFitTest[indexBestIndividual];

                tempFitnesTest = uFitTest;
                uFitTest = uFitTestNew;
                uFitTestNew = tempFitnesTest;
                tempSemanticTest = uSemanticTestCases;
                uSemanticTestCases = uSemanticTestCasesNew;
                uSemanticTestCasesNew = tempSemanticTest;
                indexBestIndividual = indexWorstOffspring;
            }else{

                tempFitnes = uFit;
                uFit = uFitNew;
                uFitNew = tempFitnes;
                tempSemantic = uSemanticTrainCases;
                uSemanticTrainCases = uSemanticTrainCasesNew;
                uSemanticTrainCasesNew = tempSemantic;
                tempFitnesTest = uFitTest;
                uFitTest = uFitTestNew;
                uFitTestNew = tempFitnesTest;
                tempSemanticTest = uSemanticTestCases;
                uSemanticTestCases = uSemanticTestCasesNew;
                uSemanticTestCasesNew = tempSemanticTest;
                indexBestIndividual = indexBestOffspring;
            }
            /*!< writing the  training fitness of the best individual on the file fitnesstrain.csv*/
            fitTraining << generation << ","<<uFit[indexBestIndividual]<<endl;
            /*!< writing the  test fitness of the best individual on the file fitnesstest.csv*/
            fitTesting << generation << ","<<uFitTest[indexBestIndividual]<<endl;
            
            hipEventRecord(stopGsgp);
            hipEventSynchronize(stopGsgp);
            hipEventElapsedTime(&generationTime, startGsgp, stopGsgp);    
        }
        hipDeviceSynchronize();
        markTracesGeneration(vectorTraces, config.populationSize, config.numberGenerations,  indexBestIndividual);
        saveTrace(outputNameFiles,logPath, vectorTraces, config.numberGenerations, config.populationSize);
            
        /*!< at the end of the execution  to deallocate memory*/
        hipFree(indexRandomTrees);
        hipFree(vectorTraces);
        hipFree(State);
        hipblasDestroy(handle);
        hipFree(dInitialPopulation);
        hipFree(dRandomTrees);
        free(hInitialPopulation);
        free(hRandomTrees);
        hipFree(uDataTrainTarget);
        hipFree(uDataTestTarget);
        hipFree(uFit);
        hipFree(uFitNew);
        hipFree(uSemanticTrainCases);
        hipFree(uSemanticRandomTrees);
        hipFree(uSemanticTrainCasesNew);
        hipFree(uSemanticTestCases);
        hipFree(uSemanticTestRandomTrees);
        hipFree(uSemanticTestCasesNew);     
        hipFree(uFitTest);
        hipFree(uFitTestNew);
        hipFree(mutationStep);
        hipEventRecord(stopRun);
        hipEventSynchronize(stopRun);
        hipEventElapsedTime(&executionTime, startRun, stopRun);

        /*!< writing the time execution for stages the algorithm*/
        times << config.populationSize
        << "," << individualLength 
        << "," << nrow 
        << "," << nvar 
        << "," << executionTime/1000
        << "," << initialitionTimePopulation/1000
        << "," << timeComputeSemantics/1000
        << "," << generationTime/1000
        <<endl;
        hipFree(states);
        hipFree(statesMutationStep);
        /*!< all device allocations are removed*/
        hipDeviceReset();
            
    }if (!la.empty()) {

        countInputFile(path_test, nrowTest, nvar);
        nvar--;

        namePopulation = la + namePopulation;
        nameRandomTrees = la + nameRandomTrees;
        std::string outFile (pathOutFile);
        outFile = logPath + outFile;
        outFile.c_str();
        individualLength = config.maxIndividualLength; /*!< Variable that stores maximum depth for individuals */
        sizeMemPopulation = sizeof(float) * config.populationSize * individualLength; /*!< Variable that stores size in bytes for initial population*/
        sizeMemIndividuals = sizeof(float) * config.populationSize; /*!< Variable that stores size in bytes of the number of individuals in the initial population*/

        float *initPopulation, *randomTress, *dInitialPopulation,*dRandomTrees; /*!< This vector pointers to store the individuals of the initial population and random trees */
        initPopulation = (float*)malloc(sizeMemPopulation); /*!<  Variable that stores the size in bytes the initial population */
        randomTress = (float*)malloc(sizeMemPopulation);  /*!< Variable that stores the size in bytes the initial population */

        checkCudaErrors(hipMalloc((void **)&dRandomTrees, sizeMemPopulation)); 
        checkCudaErrors(hipMalloc((void **)&dInitialPopulation, sizeMemPopulation));

        readPopulation(initPopulation, randomTress, config.populationSize, individualLength, logPath, namePopulation, nameRandomTrees);

        ///*!<return the initial population of the device to the host*/
        hipMemcpy(dInitialPopulation, initPopulation, sizeMemPopulation, hipMemcpyHostToDevice); 
        hipMemcpy(dRandomTrees, randomTress, sizeMemPopulation, hipMemcpyHostToDevice); 

        int sizeDataTest = sizeof(float)*(nrowTest*nvar); /*!< Variable that stores the size in bytes the size of the test data*/
        int sizeDataTestTarget = sizeof(float)*(nrowTest); /*!< Variable that stores the size in bytes the size of the target data */
        float *unssenDataTest, *dUnssenDataTest, *unssenDataTestTarget; /*!< This vector pointers to store the individuals of the test data and target data */
        unssenDataTest = (float *)malloc(sizeDataTest); /*!< Reserve memory on host*/
        unssenDataTestTarget = (float *)malloc(sizeDataTestTarget); /*!< Reserve memory on host*/
        checkCudaErrors(hipMalloc((void **)&dUnssenDataTest, sizeDataTest));

        readInpuTestData(path_test, unssenDataTest, unssenDataTestTarget, nrowTest, nvar);

        hipMemcpy(dUnssenDataTest, unssenDataTest, sizeDataTest, hipMemcpyHostToDevice); 
        
        sizeMemSemanticTest = sizeof(float)*(config.populationSize*nrowTest); /*!< Variable that stores the size in bytes of semantics for the entire population with test data*/

        checkCudaErrors(hipMalloc((void**)&uPushGenes, sizeMemIndividuals));
        checkCudaErrors(hipMalloc((void**)&uStackInd, sizeMemPopulation));  

        float *uSemanticCases, *hSemanticCases, *uSemanticRandomTrees,*hSemanticRandomTrees; /*!< pointer of vectors that contain the semantics of an individual in the population, calculated with the training set and test in generation g and its allocation in GPU*/
        checkCudaErrors(hipMalloc((void**)&uSemanticCases,sizeMemSemanticTest));            
        checkCudaErrors(hipMalloc((void**)&uSemanticRandomTrees,sizeMemSemanticTest));   
        hSemanticCases = (float*)malloc(sizeMemSemanticTest);
        hSemanticRandomTrees= (float*)malloc(sizeMemSemanticTest);             

        hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, computeSemantics, 0, config.populationSize); /*!< heuristic function used to choose a good block size is to aim at high occupancy*/
        gridSize = (config.populationSize + blockSize - 1) / blockSize; /*!< round up according to array size*/            

        /*!< invokes the GPU to interpret the initial population with data train*/
        computeSemantics<<< gridSize, blockSize >>>(dInitialPopulation, uSemanticCases, individualLength, dUnssenDataTest, nrowTest, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");
        hipMemcpy(hSemanticCases,uSemanticCases, sizeMemSemanticTest,hipMemcpyDeviceToHost);

        computeSemantics<<< gridSize, blockSize >>>(dRandomTrees, uSemanticRandomTrees, individualLength, dUnssenDataTest, nrowTest, nvar, uPushGenes, uStackInd);
        cudaErrorCheck("computeSemantics");
        hipMemcpy(hSemanticRandomTrees,uSemanticRandomTrees, sizeMemSemanticTest,hipMemcpyDeviceToHost);

        /*!< Create file for saved results of best model with the unseen data*/
        std::ofstream OUT(outFile,ios::out);
        
        evaluate_data(pathTrace, config.numberGenerations, hSemanticCases, hSemanticRandomTrees, OUT, config.logPath, nrowTest, nvar);
        
        
        free(unssenDataTest); 
        free(unssenDataTestTarget);
        free(initPopulation);
        free(randomTress);
        hipFree(dInitialPopulation);
        hipFree(dRandomTrees);
        hipFree(uSemanticCases);
        hipFree(uSemanticRandomTrees);
        hipFree(uPushGenes);
        hipFree(uStackInd);
    }
    hipDeviceReset();
    return 0;
}